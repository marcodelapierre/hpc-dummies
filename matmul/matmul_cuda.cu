#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>

#define CUDA_ERROR_CHECK(X)({\
    if((X) != hipSuccess){\
      printf("CUDA ERROR %s: %s\n", (X), hipGetErrorString((X)));\
      exit(1);\
    }\
})


// Verify MATMUL result
float verify_matmul( const float tot, const size_t n2, const float* const C )
{
  float err = 0.;
  for (size_t i = 0; i < n2; i++) {
    err += fabs( C[i] - tot );
  }

  return err;
}

// Initisalise array
__global__ void init_array( const size_t n, const float val, 
                            float* const v )
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;

  for ( int tid = index ; tid < n ; tid += stride) {
    v[tid] = val;
  }
}

// Perform MATMUL
// naive and slow
__global__ void matmul( const size_t n, const float* const A, 
            const float* const B, float* const C )
{
  int index_x = blockIdx.x * blockDim.x + threadIdx.x;
  int stride_x = blockDim.x * gridDim.x;
//  int index_y = blockIdx.y * blockDim.y + threadIdx.y;
//  int stride_y = blockDim.y * gridDim.y;

  for ( int tx = index_x ; tx < n ; tx += stride_x) {

      for ( size_t j = 0; j < n; j++ ) {
        double sum = 0;
    
        for ( size_t k = 0; k < n; k++ ) {
          double a = A[tx * n + k];
          double b = B[k * n + j];
          sum += a * b;
        }
    
        C[tx * n + j] = (float)sum;
      }

  }

}


int main( int argc, char** argv ) {

int device;
struct hipDeviceProp_t devprop;
CUDA_ERROR_CHECK(hipGetDevice( &device ));
CUDA_ERROR_CHECK(hipGetDeviceProperties( &devprop, device ));
printf("Device name: %s\n",devprop.name);

// Size of problem
const int N = 500;
const int N2 = N * N;
// Random values
const float AVAL = 30.;
const float BVAL = 40.;
const float tot = AVAL * BVAL * N;
// Allocate arrays
const size_t Nsize = N * sizeof(float);
const size_t Nsize2 = Nsize * Nsize;
float *A, *B, *C, *C_host;
CUDA_ERROR_CHECK(hipMalloc( (void**)&A, Nsize2 ));
CUDA_ERROR_CHECK(hipMalloc( (void**)&B, Nsize2 ));
CUDA_ERROR_CHECK(hipMalloc( (void**)&C, Nsize2 ));
CUDA_ERROR_CHECK(hipHostMalloc( (void**)&C_host, Nsize2 , hipHostMallocDefault, hipHostMallocDefault, hipHostMallocDefault));
// More definitions
float clocktime, err;
clock_t start, watch;

const int gridDim = 4 * devprop.multiProcessorCount;
const int blockDim = 8 * devprop.warpSize;
// Fill values
init_array<<< gridDim, blockDim >>>(N2, AVAL, A);
CUDA_ERROR_CHECK(hipGetLastError());
init_array<<< gridDim, blockDim >>>(N2, BVAL, B);
CUDA_ERROR_CHECK(hipGetLastError());
init_array<<< gridDim, blockDim >>>(N2, 0., C);
CUDA_ERROR_CHECK(hipGetLastError());
CUDA_ERROR_CHECK(hipDeviceSynchronize());

// Start timer
start = clock();

// MATMUL
// naive and slow
//const dim3 gridDim2D(32, 32); // just putting a value
//const dim3 blockDim2D(16, 16);
matmul<<< gridDim, blockDim >>>(N, A, B, C);
CUDA_ERROR_CHECK(hipGetLastError());
CUDA_ERROR_CHECK(hipDeviceSynchronize());

// Stop timer
watch = clock() - start;
clocktime = ((float)watch)/CLOCKS_PER_SEC;

// MATMUL verification
CUDA_ERROR_CHECK(hipMemcpy(C_host, C, Nsize2, hipMemcpyDeviceToHost));
err = verify_matmul( tot, N2, C_host );

// Print stuff
printf("N: %i; Err: %f; Clock[ms]: %f;\n", N, err, clocktime*1000.);

// Deallocate arrays
CUDA_ERROR_CHECK(hipHostFree( C_host ));
CUDA_ERROR_CHECK(hipFree( C ));
CUDA_ERROR_CHECK(hipFree( B ));
CUDA_ERROR_CHECK(hipFree( A ));

return 0;
}