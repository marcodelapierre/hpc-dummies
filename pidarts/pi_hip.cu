#include "hip/hip_runtime.h"
/* Compute pi in serial */
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
static long num_trials = 1000000;
 
__global__ void kernel(int* Ncirc_t_device,float *randnum)
{
  int i;
  double r = 1.0; // radius of circle
  double r2 = r*r;
  double x,y;
 
  i = blockDim.x * blockIdx.x + threadIdx.x;
  x=randnum[2*i];
  y=randnum[2*i+1];
   
  if ((x*x + y*y) <= r2)
      Ncirc_t_device[i]=1;
  else
      Ncirc_t_device[i]=0;
}
 
int main(int argc, char **argv) {
  int i;
  long Ncirc=0;
  int *Ncirc_t_device;
  int *Ncirc_t_host;
  float *randnum;
  int threads, blocks;
  double pi;
 
  // Allocate an array for the random numbers in GPU memory space
  hipMalloc((void**)&randnum,(2*num_trials)*sizeof(float));
 
  // Generate random numbers
  int status;
  hiprandGenerator_t randgen;
  status = hiprandCreateGenerator(&randgen, HIPRAND_RNG_PSEUDO_MRG32K3A);
  status |= hiprandSetPseudoRandomGeneratorSeed(randgen, 4294967296ULL^time(NULL));
  status |= hiprandGenerateUniform(randgen, randnum, (2*num_trials));
  status |= hiprandDestroyGenerator(randgen); 
 
  threads=1000;
  blocks=num_trials/threads;
 
  // Allocate hit array on host
  Ncirc_t_host=(int*)malloc(num_trials*sizeof(int));
  // Allocate hit array on device
  hipMalloc((void**)&Ncirc_t_device,num_trials*sizeof(int));
 
  hipLaunchKernelGGL(kernel, blocks, threads, 0, 0, Ncirc_t_device,randnum);
 
  // Synchronize host and device
  hipDeviceSynchronize();
 
  // Copy the hit array to host
  hipMemcpy(Ncirc_t_host,Ncirc_t_device,num_trials*sizeof(int),hipMemcpyDeviceToHost);
 
  // Count hits
  for(i=0; i<num_trials; i++)
    Ncirc+=Ncirc_t_host[i];
 
  pi = 4.0 * ((double)Ncirc)/((double)num_trials);
   
  printf("\n \t Computing pi using CUDA: \n");
  printf("\t For %ld trials, pi = %f\n", num_trials, pi);
  printf("\n");
 
  hipFree(randnum);
  hipFree(Ncirc_t_device);
  free(Ncirc_t_host);
 
  return 0;
}
