#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <hipblas.h>
#include "my_timer.h"

#define CE(op) if ((status = op) != HIPBLAS_STATUS_SUCCESS) { std::cerr << "Error: " #op << " [status=" << status << "]\n"; return 1; }


// Verify SAXPY result
float verify_saxpy( const float tot, const size_t n, const float* const y )
{
  float err = 0.;
  for (size_t i = 0; i < n; i++) {
    err += fabs( y[i] - tot );
  }

  return err;
}


int main()
{
   const unsigned N = (1 << 26);
   const float XVAL = rand() % 1000000;
   const float YVAL = rand() % 1000000;
   const float AVAL = rand() % 1000000;
   const float tot = AVAL * XVAL + YVAL;

   hipblasStatus_t status;
   hipblasHandle_t h = nullptr;
   float *host_x, *host_y;

   host_x = new float[N];
   host_y = new float[N];

   //cublasInit();
   CE( hipblasCreate(&h) );

   for (int i=0; i<N; ++i) {
      host_x[i] = XVAL;
      host_y[i] = YVAL;
   }
   float *dev_x, *dev_y;
   hipMalloc( (void**)&dev_x, N*sizeof(float));
   hipMalloc( (void**)&dev_y, N*sizeof(float));

   CE( hipblasSetVector(N, sizeof(host_x[0]), host_x, 1, dev_x, 1) );
   CE( hipblasSetVector(N, sizeof(host_y[0]), host_y, 1, dev_y, 1) );

   hipDeviceSynchronize();

   my_timer timer;
   hipblasSaxpy(h, N, &AVAL, dev_x, 1, dev_y, 1);
   hipDeviceSynchronize();
   float clocktime = (float)timer.elapsed();

   hipblasGetVector(N, sizeof(host_y[0]), dev_y, 1, host_y, 1);
   float err = verify_saxpy( tot, N, host_y );

   std::cout << "N: " << N << "; ";
   std::cout << "Err: " << err << "; ";
   std::cout << "Clock[ms]: " << clocktime*1000. << "; ";
   std::cout << std::endl;

   if (h)
      hipblasDestroy(h);
   hipFree(dev_y);
   hipFree(dev_x);
   delete [] host_y;
   delete [] host_x;

   return 0;
}
