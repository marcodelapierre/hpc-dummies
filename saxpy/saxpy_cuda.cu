#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>

#define CUDA_ERROR_CHECK(X)({\
    if((X) != hipSuccess){\
      printf("CUDA ERROR %s: %s\n", (X), hipGetErrorString((X)));\
      exit(1);\
    }\
})


// Verify SAXPY result
float verify_saxpy( const float tot, const size_t n, const float* const y )
{
  float err = 0.;
  for (size_t i = 0; i < n; i++) {
    err += fabs( y[i] - tot );
  }

  return err;
}

// Initisalise array
__global__ void init_array( const size_t n, const float val, 
                            float* const v )
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;

  for ( int tid = index ; tid < n ; tid += stride) {
    v[tid] = val;
  }
}

// Perform SAXPY
__global__ void saxpy( const size_t n, const float a, 
                       const float* const x, float* const y )
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;

  for ( int tid = index ; tid < n ; tid += stride) {
    y[tid] = a * x[tid] + y[tid];
  }
}


int main( int argc, char** argv ) {

int device;
struct hipDeviceProp_t devprop;
CUDA_ERROR_CHECK(hipGetDevice( &device ));
CUDA_ERROR_CHECK(hipGetDeviceProperties( &devprop, device ));
printf("Device name: %s\n",devprop.name);

// Size of problem
const unsigned N = (1 << 26);
// Random values
const float XVAL = rand() % 1000000;
const float YVAL = rand() % 1000000;
const float AVAL = rand() % 1000000;
const float tot = AVAL * XVAL + YVAL;
// Allocate arrays
const size_t Nsize = N * sizeof(float);
float *x, *y, *y_host;
CUDA_ERROR_CHECK(hipMalloc( (void**)&x, Nsize ));
CUDA_ERROR_CHECK(hipMalloc( (void**)&y, Nsize ));
CUDA_ERROR_CHECK(hipHostMalloc( (void**)&y_host, Nsize , hipHostMallocDefault, hipHostMallocDefault, hipHostMallocDefault));
// More definitions
float clocktime, err;
clock_t start, watch;

const int gridDim = 4 * devprop.multiProcessorCount;
const int blockDim = 8 * devprop.warpSize;
// Fill values
init_array<<< gridDim, blockDim >>>(N, XVAL, x);
CUDA_ERROR_CHECK(hipGetLastError());
init_array<<< gridDim, blockDim >>>(N, YVAL, y);
CUDA_ERROR_CHECK(hipGetLastError());
CUDA_ERROR_CHECK(hipDeviceSynchronize());

// Start timer
start = clock();

// SAXPY
saxpy<<< gridDim, blockDim >>>(N, AVAL, x, y);
CUDA_ERROR_CHECK(hipGetLastError());
CUDA_ERROR_CHECK(hipDeviceSynchronize());

// Stop timer
watch = clock() - start;
clocktime = ((float)watch)/CLOCKS_PER_SEC;

// SAXPY verification
CUDA_ERROR_CHECK(hipMemcpy(y_host, y, Nsize, hipMemcpyDeviceToHost));
err = verify_saxpy( tot, N, y_host );

// Print stuff
printf("N: %i; Err: %f; Clock[ms]: %f;\n", N, err, clocktime*1000.);

// Deallocate arrays
CUDA_ERROR_CHECK(hipHostFree( y_host ));
CUDA_ERROR_CHECK(hipFree( y ));
CUDA_ERROR_CHECK(hipFree( x ));

return 0;
}