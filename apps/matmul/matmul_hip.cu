#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>

#define CUDA_ERROR_CHECK(X)({\
    if((X) != hipSuccess){\
      printf("CUDA ERROR %s: %s\n", (X), hipGetErrorString((X)));\
      exit(1);\
    }\
})


// Verify MATMUL result
float verify_matmul( const float tot, const size_t n2, const float* const C )
{
  float err = 0.;
  for (size_t i = 0; i < n2; i++) {
    err += fabs( C[i] - tot );
  }

  return err;
}

// Initisalise array
__global__ void init_array( const size_t n, const float val, 
                            float* const v )
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;

  for ( int tid = index ; tid < n ; tid += stride) {
    v[tid] = val;
  }
}

// Perform MATMUL
__global__ void matmul( const size_t n, const float* const A, 
            const float* const B, float* const C )
{
  int index_x = blockIdx.x * blockDim.x + threadIdx.x;
  int stride_x = blockDim.x * gridDim.x;
  int index_y = blockIdx.y * blockDim.y + threadIdx.y;
  int stride_y = blockDim.y * gridDim.y;

  for ( int tx = index_x ; tx < n ; tx += stride_x) {
    for ( int ty = index_y ; ty < n ; ty += stride_y) {
      double sum = 0.;
      for ( size_t k = 0; k < n; k++ ) {
        double a = A[tx * n + k];
        double b = B[k * n + ty];
        sum += a * b;
      }
      C[tx * n + ty] = (float)sum;
    }
  }
}


int main( int argc, char** argv ) {

int device;
struct hipDeviceProp_t devprop;
CUDA_ERROR_CHECK(hipGetDevice( &device ));
CUDA_ERROR_CHECK(hipGetDeviceProperties( &devprop, device ));
printf("Device name: %s\n",devprop.name);

// Size of problem
const int N = 500;
const int N2 = N * N;
// Random values
const float AVAL = 30.;
const float BVAL = 40.;
const float tot = AVAL * BVAL * N;
// Allocate arrays
const size_t Nsize = N * sizeof(float);
const size_t Nsize2 = Nsize * Nsize;
float *A, *B, *C, *C_host;
CUDA_ERROR_CHECK(hipMalloc( (void**)&A, Nsize2 ));
CUDA_ERROR_CHECK(hipMalloc( (void**)&B, Nsize2 ));
CUDA_ERROR_CHECK(hipMalloc( (void**)&C, Nsize2 ));
CUDA_ERROR_CHECK(hipHostMalloc( (void**)&C_host, Nsize2 ));
// More definitions
float clocktime, err;
clock_t start, watch;

const int gridDim = 4 * devprop.multiProcessorCount;
const int blockDim = 8 * devprop.warpSize;
// Fill values
hipLaunchKernelGGL(init_array, dim3(gridDim), dim3(blockDim ), 0, 0, N2, AVAL, A);
CUDA_ERROR_CHECK(hipGetLastError());
hipLaunchKernelGGL(init_array, dim3(gridDim), dim3(blockDim ), 0, 0, N2, BVAL, B);
CUDA_ERROR_CHECK(hipGetLastError());
hipLaunchKernelGGL(init_array, dim3(gridDim), dim3(blockDim ), 0, 0, N2, 0., C);
CUDA_ERROR_CHECK(hipGetLastError());
CUDA_ERROR_CHECK(hipDeviceSynchronize());

// Start timer
start = clock();

// MATMUL
const dim3 gridDim2D(32, 32); // just putting a value
const dim3 blockDim2D(16, 16);
hipLaunchKernelGGL(matmul, dim3(gridDim2D), dim3(blockDim2D ), 0, 0, N, A, B, C);
CUDA_ERROR_CHECK(hipGetLastError());
CUDA_ERROR_CHECK(hipDeviceSynchronize());

// Stop timer
watch = clock() - start;
clocktime = ((float)watch)/CLOCKS_PER_SEC;

// MATMUL verification
CUDA_ERROR_CHECK(hipMemcpy(C_host, C, Nsize2, hipMemcpyDeviceToHost));
err = verify_matmul( tot, N2, C_host );

// Print stuff
printf("N: %i; Err: %f; Clock[ms]: %f;\n", N, err, clocktime*1000.);

// Deallocate arrays
CUDA_ERROR_CHECK(hipHostFree( C_host ));
CUDA_ERROR_CHECK(hipFree( C ));
CUDA_ERROR_CHECK(hipFree( B ));
CUDA_ERROR_CHECK(hipFree( A ));

return 0;
}