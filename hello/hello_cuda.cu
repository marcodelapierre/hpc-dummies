
#include <hip/hip_runtime.h>
#include <stdio.h>
 
 
__global__ void helloGPU() {
 
  int tid = threadIdx.x;
  int bid = blockIdx.x;
  printf("Hello from GPU thread %d in block %d\n",tid,bid);
 
}
 
 
int main(int argc, char *argv[]) {
 
  int no_blocks = 4;
  int no_threads = 5;
 
  helloGPU<<<no_blocks,no_threads>>>();
 
  hipDeviceSynchronize();
 
}
