#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>

#define HIP_ERROR_CHECK(X)({\
    if((X) != hipSuccess){\
      printf("HIP ERROR %s: %s\n", (X), hipGetErrorString((X)));\
      exit(1);\
    }\
})


// Verify SAXPY result
float verify_saxpy( const float tot, const size_t n, const float* const y )
{
  float err = 0;
  for (size_t i = 0; i < n; i++) {
    err += fabs( y[i] - tot );
  }

  return err;
}

// Initisalise array
__global__ void init_array( const size_t n, const float val, 
                            float* const v )
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;

  for ( int tid = index ; tid < n ; tid += stride) {
    v[tid] = val;
  }
}

// Perform SAXPY
__global__ void saxpy( const size_t n, const float a, 
                       const float* const x, float* const y )
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;

  for ( int tid = index ; tid < n ; tid += stride) {
    y[tid] = a * x[tid] + y[tid];
  }
}


int main( int argc, char** argv ) {

int device;
struct hipDeviceProp_t devprop;
HIP_ERROR_CHECK(hipGetDevice( &device ));
HIP_ERROR_CHECK(hipGetDeviceProperties( &devprop, device ));
printf("Device name: %s\n",devprop.name);

// Size of problem
const unsigned N = (1 << 26);
// Random values
const float XVAL = rand() % 1000000;
const float YVAL = rand() % 1000000;
const float AVAL = rand() % 1000000;
const float tot = AVAL * XVAL + YVAL;
// Allocate arrays
const size_t Nsize = N * sizeof(float);
float *x, *y, *y_host;
HIP_ERROR_CHECK(hipMalloc( (void**)&x, Nsize ));
HIP_ERROR_CHECK(hipMalloc( (void**)&y, Nsize ));
HIP_ERROR_CHECK(hipMallocHost( (void**)&y_host, Nsize ));
// More definitions
float clocktime, err;
clock_t start, watch;

const int gridDim = 4 * devprop.multiProcessorCount;
const int blockDim = 8 * devprop.warpSize;
// Fill values
hipLaunchKernelGGL(init_array, dim3(gridDim), dim3(blockDim ), 0, 0, N, XVAL, x);
HIP_ERROR_CHECK(hipGetLastError());
hipLaunchKernelGGL(init_array, dim3(gridDim), dim3(blockDim ), 0, 0, N, YVAL, y);
HIP_ERROR_CHECK(hipGetLastError());
HIP_ERROR_CHECK(hipDeviceSynchronize());

// Start timer
start = clock();

// SAXPY
hipLaunchKernelGGL(saxpy, dim3(gridDim), dim3(blockDim ), 0, 0, N, AVAL, x, y);
HIP_ERROR_CHECK(hipGetLastError());
HIP_ERROR_CHECK(hipDeviceSynchronize());

// Stop timer
watch = clock() - start;
clocktime = ((float)watch)/CLOCKS_PER_SEC;

// SAXPY verification
HIP_ERROR_CHECK(hipMemcpy(y_host, y, Nsize, hipMemcpyDeviceToHost));
err = verify_saxpy( tot, N, y_host );

// Print stuff
printf("N: %i; Err: %f; Clock[ms]: %f;\n", N, err, clocktime*1000.);

// Deallocate arrays
HIP_ERROR_CHECK(hipFreeHost( y_host ));
HIP_ERROR_CHECK(hipFree( y ));
HIP_ERROR_CHECK(hipFree( x ));

return 0;
}