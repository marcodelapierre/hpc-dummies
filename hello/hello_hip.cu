#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <stdio.h>
 
 
__global__ void helloGPU() {
 
  int tid = threadIdx.x;
  int bid = blockIdx.x;
  printf("Hello from GPU thread %d in block %d\n",tid,bid);
 
}
 
 
int main(int argc, char *argv[]) {
 
  int no_blocks = 4;
  int no_threads = 5;
 
  hipLaunchKernelGGL(helloGPU, dim3(no_blocks), dim3(no_threads), 0, 0);
 
  hipDeviceSynchronize();
 
}
